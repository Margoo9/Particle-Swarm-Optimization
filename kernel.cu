#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math_functions.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

hipError_t addWithCuda(double *positions, double *velocities, double *personalBests,
	double *globalBest);

#define M_PI 3.14159265358979323846

const int particlesNum = 1024;
const int dimensionsNum = 2;
const int iterationsNum = 2000;


double beale_function_h(double *x);
double easome_function_h(double *x);
double rosenbrock_function_h(double *x);


__device__ double beale_function(double *x);
__device__ double easome_function(double *x);
__device__ double rosenbrock_function(double *x);


__global__ void psoKernel(double *positions, double *velocities,
	double *personalBests, double *globalBest, double r1,
	double r2)
{

	const double c1 = 2;
	const double c2 = 2;
	const double w = 0.5;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = i % dimensionsNum;

	if (i >= particlesNum * dimensionsNum)
		return;

	//vel= inertia component + cognitive component + social component
	velocities[i] = w * velocities[i] + c1 * r1 * (personalBests[i] - positions[i])
		+ c2 * r2 * (globalBest[tid] - positions[i]);

	//updated position
	positions[i] += velocities[i];


	//finding personal best
	double temp1[dimensionsNum];
	double temp2[dimensionsNum];

	if (tid != 0)
		return;

	for (int j = 0; j < dimensionsNum; j++)
    {
        temp1[j] = positions[i + j];
    }

    for (int j = 0; j < dimensionsNum; j++)
    {
        temp2[j] = personalBests[i + j];
    }
	
	if (easome_function(temp2) > easome_function(temp1))
	{
		for (int j = 0; j < dimensionsNum; j++)
			personalBests[i + j] = positions[i + j];
	}
}

void globalBestFunction(double *personalBests, double *globalBest) {

	double temp[dimensionsNum];
	for (int i = 0; i < particlesNum * dimensionsNum; i += dimensionsNum)
	{
		for (int k = 0; k < dimensionsNum; k++)
			temp[k] = personalBests[i + k];

		if (easome_function_h(globalBest) > easome_function_h(temp))
		{
			for (int k = 0; k < dimensionsNum; k++)
				globalBest[k] = temp[k];
		}
	}
}


int main()
{

	double positions[particlesNum * dimensionsNum];
	double velocities[particlesNum * dimensionsNum];
	double personalBests[particlesNum * dimensionsNum];
	double globalBest[dimensionsNum];


	for (int i = 0; i < particlesNum * dimensionsNum; i++)
	{
		positions[i] = (-4.5f) + double(((4.5f - (-4.5f)) + 1) * rand() / (RAND_MAX + 1.0));
		personalBests[i] = positions[i];
		velocities[i] = 0;
	}

	for (int i = 0; i < dimensionsNum; i++)
		globalBest[i] = personalBests[i];

	clock_t start = clock();

	hipError_t cudaStatus = addWithCuda(positions, velocities, personalBests, globalBest);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}


	for (int i = 0; i < dimensionsNum; i++)
		printf("x%d = %f\n", i, globalBest[i]);

	printf("Minimum = %f", easome_function_h(globalBest));
	printf("\n");

	clock_t stop = clock();

	printf("Computing time: %f ms\n",
		(double)(stop - start) / CLOCKS_PER_SEC);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


double beale_function_h(double *x)
{
	double res = 0;

	//res += pow((1.5-x[0]+x[0]*x[0]), 2) + pow((2.25-x[0]+x[0]*x[1]*x[1]), 2) + pow((2.625-x[0]+x[0]*x[1]*x[1]*x[1]), 2);


	for (int i = 0; i < dimensionsNum - 1; i++)
	{
		double y = x[i];
		double y1 = x[i + 1];

		res += pow((1.5 - y + y * y1), 2) + pow((2.25 - y + y * y1*y1), 2) + pow((2.625 - y + y * y1*y1*y1), 2);
	}

	return res;
}

double easome_function_h(double *x)
{
	double res = 0;

	//res = -cos(x[0])*cos(x[1])*exp((-pow( x[0]-M_PI, 2)) - pow(x[1]-M_PI, 2));


	for (int i = 0; i < dimensionsNum - 1; i++)
	{
		double y = x[i];
		double y1 = x[i + 1];

		res += -cos(y)*cos(y1)*exp((-pow((y - M_PI), 2)) - pow((y1 - M_PI), 2));
	}

	return res;
}

double rosenbrock_function_h(double *x)
{

	double res = 0.0;
	double sum = 0.0;

	//res = -cos(x[0])*cos(x[1])*exp((-pow( x[0]-M_PI, 2)) - pow(x[1]-M_PI, 2));


	for (int i = 0; i < dimensionsNum - 1; i++)
	{
		double y = x[i];
		double y1 = x[i + 1];

		res += 100 * pow((y1 - (y*y)), 2) + pow((y - 1), 2);
	}

	return res;
}


__device__ double beale_function(double *x)
{

	double res = 0;
	double yn = x[dimensionsNum - 1];

	// res += pow((1.5-x[0]+x[0]*x[0]), 2) + pow((2.25-x[0]+x[0]*x[1]*x[1]), 2) + pow((2.625-x[0]+x[0]*x[1]*x[1]*x[1]), 2);

	for (int i = 0; i < dimensionsNum - 1; i++)
	{
		double y = x[i];
		double y1 = x[i + 1];

		res += pow((1.5 - y + y * y1), 2) + pow((2.25 - y + y * y1*y1), 2) + pow((2.625 - y + y * y1*y1*y1), 2);
	}

	return res;
}

__device__ double easome_function(double *x)
{

	double res = 0;

	//res = -cos(x[0])*cos(x[1])*exp((-pow( x[0]-M_PI, 2)) - pow(x[1]-M_PI, 2));


	for (int i = 0; i < dimensionsNum - 1; i++)
	{
		double y = x[i];
		double y1 = x[i + 1];

		res += -cos(y)*cos(y1)*exp((-pow((y - M_PI), 2)) - pow((y1 - M_PI), 2));
	}

	return res;
}


__device__ double rosenbrock_function(double *x)
{

	double res = 0.0;

	//res = -cos(x[0])*cos(x[1])*exp((-pow( x[0]-M_PI, 2)) - pow(x[1]-M_PI, 2));


	for (int i = 0; i < dimensionsNum - 1; i++)
	{
		double y = x[i];
		double y1 = x[i + 1];

		res += 100 * pow((y1 - (y*y)), 2) + pow((y - 1), 2);
	}

	return res;
}



// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *positions, double *velocities, double *personalBests,
	double *globalBest)
{
	int SIZE = particlesNum * dimensionsNum;

	double *dev_positions;
	double *dev_velocity;
	double *dev_particleBest;
	double *dev_gloabalBest;

	//double temp[dimensionsNum];

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_positions, SIZE * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_velocity, SIZE * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_particleBest, SIZE * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&dev_gloabalBest, sizeof(double) * dimensionsNum);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_positions, positions, SIZE * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_velocity, velocities, SIZE * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_particleBest, personalBests, SIZE * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_gloabalBest, globalBest, sizeof(double) * dimensionsNum, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.


	int threadsNum = 64;
	int blocksNum = particlesNum / threadsNum;

	for (int i = 0; i < iterationsNum; i++)
	{
		psoKernel << <blocksNum, threadsNum >> > (dev_positions, dev_velocity,
			dev_particleBest, dev_gloabalBest,
			(double)rand() / (double)RAND_MAX,
			(double)rand() / (double)RAND_MAX);



		cudaStatus = hipMemcpy(personalBests, dev_particleBest, sizeof(double) * particlesNum * dimensionsNum, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		globalBestFunction(personalBests, globalBest);

		cudaStatus = hipMemcpy(dev_gloabalBest, globalBest, sizeof(double) * dimensionsNum, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
	}

	cudaStatus = hipMemcpy(positions, dev_positions, SIZE * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(velocities, dev_velocity, SIZE * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(personalBests, dev_particleBest, SIZE * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(globalBest, dev_gloabalBest, sizeof(double) * dimensionsNum, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


Error:
	hipFree(dev_positions);
	hipFree(dev_velocity);
	hipFree(dev_particleBest);
	hipFree(dev_gloabalBest);

	return cudaStatus;
}